#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void vectorAdd(const float *A, const float *B, float *C, int numElements) {
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if (i < numElements) {
        C[i] = A[i] + B[i];
    }
}

extern "C" void gpu_alloc(void** devicePtr, size_t size) {
    hipMalloc((void**)devicePtr, size);
}

extern "C" void host_to_gpu(void* dst, const void* src, size_t count) {
    hipMemcpy(dst, src, count, hipMemcpyHostToDevice);
}

extern "C" void gpu_to_host(void* dst, const void* src, size_t count) {
    hipMemcpy(dst, src, count, hipMemcpyDeviceToHost);
}

extern "C" void gpu_free(void* devicePtr) {
    hipFree(devicePtr);
}

extern "C" void gpu_run(const float *A, const float *B, float *C, int N) {
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(A, B, C, N);
    hipDeviceSynchronize();
}